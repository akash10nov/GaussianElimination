
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <unistd.h>
#include <sys/time.h>
#include <stdint.h>
#define MAX 10
#define MIN 1

void lu_decomp(float *a, float *u,int dimension);
__global__ void DUKernel(float *D_a, float *D_u,unsigned int size);
uint64_t getTime();

int main(int argc, char **argv){	
	float *a, *u, *l;
	int dimension;
	
	dimension = atoi(argv[1]);
	a= (float*)malloc(sizeof(float) * (dimension*dimension));
	l= (float*)malloc(sizeof(float) * (dimension*dimension));
	u= (float*)malloc(sizeof(float) * (dimension*dimension));
	for(int i = 0; i<dimension; i++)
	 {	 
       for(int j = 0; j<dimension; j++)
	   {	   
            a[(i*dimension)+j] = rand() % (MAX - MIN) + MIN;
		   u[(i*dimension)+j] =  a[(i*dimension)+j];
		  
		  if(i == j)
		  {
			 l[(i*dimension)+j] = 1;  
		  }
		  else
		  {
			 l[(i*dimension)+j] = 0;  
		  }		
	   } 
	 }  
	 for(int k = 0; k < dimension-1; k++)
	 {
		for(int j=k+1; j < dimension; j++ )
		{
			l[(j*dimension)+k] = a[(j*dimension)+k]/a[(k*dimension)+k];
			u[(j*dimension)+k]=0;
		}
	}
	/*printf("U before\n");
	for(int i = 0; i<dimension; i++)
	 {		
	 for(int j = 0; j<dimension; j++)
	   {	   
            printf("%15f",u[(i*dimension)+j]);
	   }
	   printf("\n");
	 }*/
	
      
	lu_decomp(a,u,dimension); 
	

	/*
	remove this comment for verification part

	
	float temp =0;
	float x=0;
	float diff_allowed=10;
	for(int i =0; i < dimension; i++)
	{
		for(int j=0; j < dimension; j++)
		{
			temp =0;
			for(int k=0; k < dimension; k++)
			{
			   temp = temp + l[(i*dimension)+k]* u[(k*dimension)+j];
			   temp=a[(i*dimension)+j];	
			}
			
			//printf("%15f",temp);
			
		   	 if((abs(temp-a[(i*dimension)+j])>diff_allowed))   
			{
				x=abs(temp-a[(i*dimension)+j]);
				printf("problem");
				printf("diff: %5f\n",x);
			}		
		}	
		//printf("\n");
	}









	remove this comment for verification


	*/
	//printf("\n");
        //printf("U Matrix:\n");
	/*
	for(int i = 0; i<dimension; i++)
	 {		
	 for(int j = 0; j<dimension; j++)
	   {	   
            printf("%15f",u[(i*dimension)+j]);
	   }
	   printf("\n");
	 }
	
	for(int i = 0; i<dimension; i++)
	 {		
	 for(int j = 0; j<dimension; j++)
	   {	   
            printf("%15f",l[(i*dimension)+j]);
	   }
	   printf("\n");
	 }
	 printf("\n");
	 printf("Original Matrix:\n");

	for(int i = 0; i<dimension; i++)
	 {		
	 for(int j = 0; j<dimension; j++)
	   {	   
            printf("%15f",a[(i*dimension)+j]);
	   }
	   printf("\n");
	 }*/
	return 0;
}

void lu_decomp(float *a,float *u, int dimension) 
{ 
    float *d_a ,*d_u; 
    uint64_t astart, aend;
     astart = getTime();
    hipMalloc(&d_a, (dimension*dimension)*sizeof(float));
	hipMalloc(&d_u, (dimension*dimension)*sizeof(float));
   
    //Copying data to device from host 
    hipMemcpy(d_a, a, sizeof(float)*dimension*(dimension),hipMemcpyHostToDevice);
	hipMemcpy(d_u, u, sizeof(float)*dimension*(dimension),hipMemcpyHostToDevice);
    
    //Kernel call 
    if(dimension<1001)
   	 DUKernel<<<dimension ,dimension,4*dimension*dimension>>>(d_a, d_u ,dimension); 
    else
	 DUKernel<<<(dimension*dimension/1000),1000,4*dimension*dimension>>>(d_a, d_u ,dimension); 
    //DUKernel<<<1024 ,100,4*dimension*dimension>>>(d_a,d_u, dimension); 
    //Coping data to host from device 
    hipMemcpy(a,d_a,sizeof(float)*dimension*(dimension),hipMemcpyDeviceToHost);
	//cudaMemcpy(l,d_l,sizeof(float)*dimension*(dimension),cudaMemcpyDeviceToHost);
	hipMemcpy(u,d_u,sizeof(float)*dimension*(dimension),hipMemcpyDeviceToHost);
	
    //Deallocating memory on the device 
    hipFree(d_a); 
    hipFree(d_u);
    aend = getTime();
	 printf("%d ,%f \n",dimension,(aend-astart)/1000000.0); 
}


__global__ void DUKernel(float *D_a,float *D_u, unsigned int dimension)
{
	// 10x10 size matrix is for experiment, so argv[1]=10
 
   	 extern __shared__ float temp[];	 
     	 int k=threadIdx.x;
	 int j=blockIdx.x;
	 int p= threadIdx.x+(blockIdx.x*blockDim.x);
	 temp[p]=D_u[p];
	 __syncthreads();
	int i=0;
	int s=0;
	 while(i<threadIdx.x && s< blockIdx.x)
	 {
		temp[p]=temp[p]-(temp[(s*dimension)+(k*(j/1000))+k] * ((temp[(j*dimension)+(i*(j/1000))+i])/temp[(j*dimension)+(j*(j/1000))+j]));
		i++;
		s++;
	 }
	/* printf("outside1 Temp:%10f k:%d j:%d\n",temp[(k*dimension)+j],k,j);

		float p=temp[(j*dimension)+k]/temp[(k*dimension)+k];		
		for(int i=(k+1);i<dimension;i++)
		{	
			
			//printf("inside loop%d\n",i);
			//printf("before Temp:%10f,j:%d i:%d\n",temp[(j*dimension)+i]);
			temp[(j*dimension)+i]=temp[(j*dimension)+i]-(temp[(k*dimension)+i]*p);
			//printf("after:Temp:%10f\n",temp[j*dimension+i]);
			//printf("after j:%d i:%d",j,i);
			
		}*/
	 __syncthreads();
	 D_u[p]=temp[p];
	
	
	
}
	


uint64_t getTime(){
	struct timeval t;
	gettimeofday(&t, NULL);
	return (uint64_t)(t.tv_sec)*1000000 + (uint64_t)(t.tv_usec);
}