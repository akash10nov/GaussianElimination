
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <unistd.h>
#include <sys/time.h>
#include <stdint.h>
#define MAX 10
#define MIN 1

void lu_decomp(float *a, float *u,int dimension);
__global__ void DUKernel(float *D_a, float *D_u,unsigned int size);
uint64_t getTime();

int main(int argc, char **argv){	
	float *a, *u, *l;
	int dimension;
	
	dimension = atoi(argv[1]);
	a= (float*)malloc(sizeof(float) * (dimension*dimension));
	l= (float*)malloc(sizeof(float) * (dimension*dimension));
	u= (float*)malloc(sizeof(float) * (dimension*dimension));
	for(int i = 0; i<dimension; i++)
	 {	 
       for(int j = 0; j<dimension; j++)
	   {	   
            a[(i*dimension)+j] = rand() % (MAX - MIN) + MIN;
		   u[(i*dimension)+j] =  a[(i*dimension)+j];
		  
		  if(i == j)
		  {
			 l[(i*dimension)+j] = 1;  
		  }
		  else
		  {
			 l[(i*dimension)+j] = 0;  
		  }		
	   } 
	 }  
	 for(int k = 0; k < dimension-1; k++)
	 {
		for(int j=k+1; j < dimension; j++ )
		{
			l[(j*dimension)+k] = a[(j*dimension)+k]/a[(k*dimension)+k];
			u[(j*dimension)+k]=0;
		}
	}
	/*printf("U before\n");
	for(int i = 0; i<dimension; i++)
	 {		
	 for(int j = 0; j<dimension; j++)
	   {	   
            printf("%15f",u[(i*dimension)+j]);
	   }
	   printf("\n");
	 }*/
	float diff_allowed=10;
      
	lu_decomp(a,u,dimension); 	
	float temp =0;
	float x=0;


	/* remove this comment for verification





	for(int i =0; i < dimension; i++)
	{
		for(int j=0; j < dimension; j++)
		{
			temp =0;
			for(int k=0; k < dimension; k++)
			{
			   temp = temp + l[(i*dimension)+k]* u[(k*dimension)+j];
			   temp=a[(i*dimension)+j];	
			}
			
			//printf("%15f",temp);
			
		   	 if((abs(temp-a[(i*dimension)+j])>diff_allowed))   
			{
				x=abs(temp-a[(i*dimension)+j]);
				printf("problem");
				printf("diff: %5f\n",x);
			}		
		}	
		//printf("\n");
	}
	


	REMOVE THIS COMMENT FOR VERIFICATION.

	*/ 
	//printf("\n");
        //printf("U Matrix:\n");
	/*
	for(int i = 0; i<dimension; i++)
	 {		
	 for(int j = 0; j<dimension; j++)
	   {	   
            printf("%15f",u[(i*dimension)+j]);
	   }
	   printf("\n");
	 }
	
	for(int i = 0; i<dimension; i++)
	 {		
	 for(int j = 0; j<dimension; j++)
	   {	   
            printf("%15f",l[(i*dimension)+j]);
	   }
	   printf("\n");
	 }
	 printf("\n");
	 printf("Original Matrix:\n");

	for(int i = 0; i<dimension; i++)
	 {		
	 for(int j = 0; j<dimension; j++)
	   {	   
            printf("%15f",a[(i*dimension)+j]);
	   }
	   printf("\n");
	 }*/
	return 0;
}

void lu_decomp(float *a,float *u, int dimension) 
{ 
    float *d_a ,*d_u; 
    uint64_t astart, aend;
    hipMalloc(&d_a, (dimension*dimension)*sizeof(float));
	hipMalloc(&d_u, (dimension*dimension)*sizeof(float));
    astart = getTime();
    //Copying data to device from host 
    hipMemcpy(d_a, a, sizeof(float)*dimension*(dimension),hipMemcpyHostToDevice);
	hipMemcpy(d_u, u, sizeof(float)*dimension*(dimension),hipMemcpyHostToDevice);
    
    //Kernel call 
    if(dimension<1001)
   	 DUKernel<<<dimension ,dimension>>>(d_a, d_u ,dimension); 
    else
	 DUKernel<<<(dimension*dimension/1000),1000>>>(d_a, d_u ,dimension); 
    //DUKernel<<<1024 ,100,4*dimension*dimension>>>(d_a,d_u, dimension); 
    //Coping data to host from device 
    //cudaMemcpy(a,d_a,sizeof(float)*dimension*(dimension),cudaMemcpyDeviceToHost);
	//cudaMemcpy(l,d_l,sizeof(float)*dimension*(dimension),cudaMemcpyDeviceToHost);
	hipMemcpy(u,d_u,sizeof(float)*dimension*(dimension),hipMemcpyDeviceToHost);
	aend = getTime();
	 printf("%d, %f  \n",dimension,(aend-astart)/1000000.0);
    //Deallocating memory on the device 
    hipFree(d_a); 
    hipFree(d_u); 
}


__global__ void DUKernel(float *D_a,float *D_u, unsigned int dimension)
{
	// 10x10 size matrix is for experiment, so argv[1]=10
	 
	 int k=threadIdx.x;
	 int j=blockIdx.x;
	 int p= threadIdx.x+(blockIdx.x*blockDim.x);
	 __syncthreads();
	int i=0;
	int s=0;
	 while(i<threadIdx.x && s< blockIdx.x)
	 {
		D_u[p]=D_u[p]-(D_u[((s%1000)*dimension)+(k*(j/1000))+k] * ((D_u[((j%1000)*dimension)+(i*(j/1000))+i])/D_u[((j%1000)*dimension)+(j*(j/1000))+j]));
		i++;
		s++;
	 }
	// __syncthreads();
	
	
	
	
}
	


uint64_t getTime(){
	struct timeval t;
	gettimeofday(&t, NULL);
	return (uint64_t)(t.tv_sec)*1000000 + (uint64_t)(t.tv_usec);
}